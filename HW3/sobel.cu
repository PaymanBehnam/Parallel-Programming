#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include "string.h"
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define DEFAULT_THRESHOLD  4000

#define DEFAULT_FILENAME "BWstop-sign.ppm"

#define cudaCheck(x) _cudaCheck(x, #x ,__FILE__, __LINE__)
#define ThreadNumberPB 1024

template<typename T>
void _cudaCheck(T e, const char* func, const char* call, const int line){
  if(e != hipSuccess){
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

//////////////////////
///        START_GPU
//////////////////////
__global__ void sobel_gpu(unsigned int *imgin, unsigned int *imgout, int width, int height) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int sum1, sum2, magnitude;
    int offset = y*width + x;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        sum1 = (-1* imgin[(y-1)*width + (x-1)]) + (-2*imgin[y*width+(x-1)]) + (-1*imgin[(y+1)*width+(x-1)]) +
             (imgin[(y-1)*width + (x+1)]) + ( 2*imgin[y*width+(x+1)]) + (imgin[(y+1)*width+(x+1)]);
        sum2 = (imgin[(y-1)*width + (x-1)]) + ( 2*imgin[(y-1)*width+x]) + (imgin[(y-1)*width+(x+1)]) +
             (-1* imgin[(y+1)*width + (x-1)]) + (-2*imgin[(y+1)*width+x]) + (-1*imgin[(y+1)*width+(x+1)]);
         magnitude =  (sum1*sum1) + (sum2*sum2);
        if (magnitude > DEFAULT_THRESHOLD )
          imgout[offset] = 255;
        else
          imgout[offset] = 0;
    }

}
//////////////////////
///        END_GPU
//////////////////////


unsigned int *read_ppm( char *filename, int & xsize, int & ysize, int & maxval ){

  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  fprintf(stderr, "read_ppm( %s )\n", filename);
  int fd = open( filename, O_RDONLY);
  if (fd == -1)
    {
      fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
      return NULL; // fail

    }

  char chars[1024];
  int num = read(fd, chars, 1000);

  if (chars[0] != 'P' || chars[1] != '6')
    {
      fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
      return NULL;
    }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line!
    {
      ptr = 1 + strstr(ptr, "\n");
    }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);
  xsize = width;
  ysize = height;
  maxval = maxvalue;

  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if (maxval > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }





  // TODO really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data.   no doubt stupid
  sprintf(duh, "%d\0", xsize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", ysize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", maxval);
  line = strstr(line, duh);


  fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  lseek(fd, offset, SEEK_SET); // move to the correct offset
  long numread = read(fd, buf, bufsize);
  fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize);

  close(fd);


  int pixels = xsize * ysize;
  for (int i=0; i<pixels; i++) pic[i] = (int) buf[3*i];  // red channel



  return pic; // success
}












void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic)
{

  FILE *fp;

  fp = fopen(filename, "w");
  if (!fp)
    {
      fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
      exit(-1);
    }
  //int x,y;


  fprintf(fp, "P6\n");
  fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);

  int numpix = xsize * ysize;
  for (int i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc);
  }
  fclose(fp);

}
//////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
main( int argc, char **argv )
{

  int thresh = DEFAULT_THRESHOLD;
  char *filename;
  filename = strdup( DEFAULT_FILENAME);

  if (argc > 1) {
    if (argc == 3)  { // filename AND threshold
      filename = strdup( argv[1]);
       thresh = atoi( argv[2] );
    }
    if (argc == 2) { // default file but specified threshhold

      thresh = atoi( argv[1] );
    }

    fprintf(stderr, "file %s    threshold %d\n", filename, thresh);
  }


  int xsize, ysize, maxval;
  unsigned int *pic = read_ppm( filename, xsize, ysize, maxval );


  int numbytes =  xsize * ysize * 3 * sizeof( int );
  int *result = (int *) malloc( numbytes );
  int *resultgpu = (int *) malloc( numbytes );
  if (!result) {
    fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
    exit(-1); // fail
  }

  int i, j, magnitude, sum1, sum2;
  int *out = result;

  for (int col=0; col<ysize; col++) {
    for (int row=0; row<xsize; row++) {
      *out++ = 0;
    }
  }

  for (i = 1;  i < ysize - 1; i++) {
    for (j = 1; j < xsize -1; j++) {

      int offset = i*xsize + j;

      sum1 =  pic[ xsize * (i-1) + j+1 ] -     pic[ xsize*(i-1) + j-1 ]
        + 2 * pic[ xsize * (i)   + j+1 ] - 2 * pic[ xsize*(i)   + j-1 ]
        +     pic[ xsize * (i+1) + j+1 ] -     pic[ xsize*(i+1) + j-1 ];

      sum2 = pic[ xsize * (i-1) + j-1 ] + 2 * pic[ xsize * (i-1) + j ]  + pic[ xsize * (i-1) + j+1 ]
            - pic[xsize * (i+1) + j-1 ] - 2 * pic[ xsize * (i+1) + j ] - pic[ xsize * (i+1) + j+1 ];

      magnitude =  sum1*sum1 + sum2*sum2;

      if (magnitude > thresh)
        result[offset] = 255;
      else
        result[offset] = 0;
    }
  }

  write_ppm( "resultCPU.ppm", xsize, ysize, 255, result);

  fprintf(stderr, "sobel CPU done\n");





  /////////////
  unsigned int *d_pic, *d_result;

  hipMalloc( (void**) &d_pic, numbytes);

  hipMalloc( (void**) &d_result, numbytes);

  /** Transfer over the memory from host to device and memset the sobel array to 0s **/
  hipMemcpy(d_pic, pic, numbytes/3, hipMemcpyHostToDevice);
  //hipMemset(gpu_sobel, 0, (origImg.width*origImg.height));


  // Timing using cudaEvent
  hipEvent_t start, stop;
  float et;
  cudaCheck(hipEventCreate(&start));
  cudaCheck(hipEventCreate(&stop));

	//num_threads = (sz+nelem-1)/nelem;
  dim3 numThreads(32, 32, 1);
  dim3 numBlocks(ceil(xsize/32), ceil(ysize/32), 1);


  // Time event start
  cudaCheck(hipEventRecord(start));

  {
    // TODO Invoke the kernel code here
    sobel_gpu<<<numBlocks, numThreads>>>(d_pic, d_result, xsize, ysize);


  }

  cudaCheck(hipGetLastError());

  // Time event end
  cudaCheck(hipEventRecord(stop));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&et, start, stop));
  cudaCheck(hipEventDestroy(start));
  cudaCheck(hipEventDestroy(stop));

  fprintf(stderr, "GPUTIME:");
  fprintf(stderr,"\t%0.3f\n", et);

  /** Copy data back to CPU from GPU **/
  hipMemcpy(resultgpu, d_result, numbytes, hipMemcpyDeviceToHost);
  hipFree (d_pic);
  hipFree (d_result);


  write_ppm( "resultgpu.ppm", xsize, ysize, 255, resultgpu);

  fprintf(stderr, "sobel gpu done\n");
/////////////////////////
  int index1, index2;
  for (index1 = 1; index1 < ysize-1; index1++) {
    for (index2 = 1; index2 < xsize-1; index2++) {
    if (fabsf(result[index1*xsize+index2] - resultgpu[index1*xsize+index2]) > 1e-5) {
      fprintf(stderr, "comparsion fails\n");
      return 0;
    }

  }
}
  fprintf(stderr, "comparsion passed\n");
  return 1;
}
